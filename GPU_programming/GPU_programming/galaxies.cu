
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <fstream>
#include <math.h>
using namespace std;

const int blockSize = 32;
const int GALAXY_COUNT = 200000;
const float pi = 3.141592653589;
const float conversionFactor = (float)1 / (float)60 * pi / (float)180;

__global__ void calculateAngles(float* d_realGalaxies, float* d_syntheticGalaxies, float* d_dotProducts) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int z = 0;
	int y = 0;
	for (int i = index; i < 10000000000; i += stride)
	{
		for (int j = 0; j < 100000; j = j + 2) {
			d_dotProducts[z] = d_realGalaxies[y] * d_realGalaxies[j + 1] + d_realGalaxies[y + 1] * d_realGalaxies[j];
			if (d_dotProducts[z] < -1) {
				d_dotProducts[z] == -1;
			}
			if (d_dotProducts[z] > 1) {
				d_dotProducts[z] == 1;
			}
			++z;
			y = y + 2;
		}
	}
}

float* readFile(char* filename) {
	ifstream file;
	int i = 0;
	float coordinates;
	float* galaxies = new float[GALAXY_COUNT];
	file.open(filename);
	if (!file) {
		printf("Error opening the file");
		exit(1);
	}
	while (file >> coordinates)
	{
		// Converting from arch minutes to radians.
		galaxies[i] = coordinates * conversionFactor;
		++i;
	}
	file.close();
	return galaxies;
}

int main() {
	printf("%f", conversionFactor);
	int arraySize = (GALAXY_COUNT / 2) * sizeof(float);
	float* galaxyAngles = new float[arraySize * arraySize];
	float* h_realGalaxies = readFile("data_100k_arcmin.txt");
	float* h_syntheticGalaxies = readFile("flat_100k_arcmin.txt");
	float* h_dotProducts = new float[arraySize * arraySize];
	float* d_realGalaxies; hipMalloc(&d_realGalaxies, arraySize);
	float* d_syntheticGalaxies; hipMalloc(&d_syntheticGalaxies, arraySize);
	float* d_dotProducts; hipMalloc(&d_dotProducts, arraySize);
	// Intializing the CUDA computation kernel
	int threadsInBlock = 256;
	int blocksInGrid = 100;
	hipMemcpy(d_realGalaxies, h_realGalaxies, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_syntheticGalaxies, h_syntheticGalaxies, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_dotProducts, h_dotProducts, arraySize, hipMemcpyHostToDevice);
	// Execute the function on GPU
	calculateAngles <<<blocksInGrid, threadsInBlock>>> (d_realGalaxies, d_syntheticGalaxies, d_dotProducts);
	hipMemcpy(h_dotProducts, d_dotProducts, arraySize, hipMemcpyDeviceToHost);
	for (int i = 0; i < 1000000; i++)
	{
		galaxyAngles[i] = h_dotProducts[i];
	}
	for (int i = 0; i < 1000000; i++)
	{
		printf("%f \n", galaxyAngles[i]);
	}
}