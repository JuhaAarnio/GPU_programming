
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <fstream>
#include <vector>
#include <tuple>
using namespace std;

const int blockSize = 32;
const int GALAXY_COUNT = 200000;

__global__ void calculateAngles(float* d_realGalaxies, float* d_syntheticGalaxies, float* d_dotProducts) {
	float dotProduct = 0;
	for (int i = 0; i < 25000; ++i)
	{
		dotProduct = d_realGalaxies[i] * d_realGalaxies[i + 2] + d_realGalaxies[i + 1] * d_realGalaxies[i + 3];
		d_dotProducts[i] = dotProduct;
	} 
}

float* readFile(char* filename) {
	ifstream file;
	int i = 0;
	float coordinates;
	float* galaxies = new float[GALAXY_COUNT];
	file.open(filename);
	if (!file) {
		printf("Error opening the file");
		exit(1);
	}
	while (file >> coordinates)
	{
		galaxies[i] = coordinates;
		++i;
	}
	file.close();
	return galaxies;
}

int main() {
	int arraySize = (GALAXY_COUNT / 2) * sizeof(float);
	float* h_realGalaxies = readFile("data_100k_arcmin.txt");
	float* h_syntheticGalaxies = readFile("flat_100k_arcmin.txt");
	float* h_dotProducts = new float[arraySize];
	float* d_realGalaxies; hipMalloc(&d_realGalaxies, arraySize);
	float* d_syntheticGalaxies; hipMalloc(&d_syntheticGalaxies, arraySize);
	float* d_dotProducts; hipMalloc(&d_dotProducts, arraySize);
	// Intializing the CUDA computation kernel
	/*dim3 blockDimension(blockSize, 1);
	dim3 gridDimension(1, 1);*/
	int threadsInBlock = 1024;
	int blocksInGrid = (blockSize + threadsInBlock - 1) / threadsInBlock;
	hipMemcpy(d_realGalaxies, h_realGalaxies, 100000 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_syntheticGalaxies, h_syntheticGalaxies, 100000 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dotProducts, h_dotProducts, 100000 * sizeof(float), hipMemcpyHostToDevice);
	// Execute the function on GPU
	calculateAngles <<<blocksInGrid, threadsInBlock>>> (d_realGalaxies, d_syntheticGalaxies, d_dotProducts);
	hipMemcpy(h_dotProducts, d_dotProducts, arraySize, hipMemcpyDeviceToHost);
	for (int i = 0; i < 25000; i++)
	{
		printf("%f \n", h_dotProducts[i]);
	}
}