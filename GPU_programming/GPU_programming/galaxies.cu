
#include "hip/hip_runtime.h"


#include <stdio.h>

const int blockSize = 32;
const int N = 32;
const int GALAXY_COUNT = 100000;

__global__;
void calculateAngles(char* a, int* b) {

}

void readFile(char* filename) {
    int i = 0;
    FILE* fp;
    char* coordinates;
    char* galaxies[GALAXY_COUNT];
    fp = fopen(filename, "r");
    if (fp == NULL) {
        printf("Unable to open the file");
    }
    while (fgets(coordinates, GALAXY_COUNT, fp) != NULL) {
        galaxies[i] = coordinates;
        i++;
    }
}