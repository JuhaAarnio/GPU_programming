#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>
#include <math.h>
using namespace std;

const int blockSize = 32;
const int GALAXY_COUNT = 200000;
const float pi = 3.141592653589;
const float conversionFactor = (float)1 / (float)60 * pi / (float)180;

__global__ void calculateAngles(float* d_realGalaxies, float* d_syntheticGalaxies, float* d_RdotProducts, float* d_SdotProducts, float* d_RSdotProducts) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int z = 0;
	int y = 0;
	for (int i = index; i < 1000000; i += stride)
	{
		for (int j = 0; j < 1000; j = j + 2) {
			d_RdotProducts[z] = d_realGalaxies[y] * d_realGalaxies[j + 1] + d_realGalaxies[y + 1] * d_realGalaxies[j];
			if (d_RdotProducts[z] < -1) {
				d_RdotProducts[z] = -1;
			}
			if (d_RdotProducts[z] > 1) {
				d_RdotProducts[z] = 1;
			}
			d_SdotProducts[z] = d_syntheticGalaxies[y] * d_syntheticGalaxies[j + 1] + d_syntheticGalaxies[y + 1] * d_syntheticGalaxies[j];
			if (d_SdotProducts[z] < -1) {
				d_SdotProducts[z] = -1;
			}
			if (d_SdotProducts[z] > 1) {
				d_SdotProducts[z] = 1;
			}
			d_RSdotProducts[z] = d_realGalaxies[y] * d_syntheticGalaxies[j + 1] + d_realGalaxies[y + 1] * d_syntheticGalaxies[j];
			if (d_RSdotProducts[z] < -1) {
				d_RSdotProducts[z] = -1;
			}
			if (d_RSdotProducts[z] > 1) {
				d_RSdotProducts[z] = 1;
			}
			++z;
			y = y + 2;
		}
	}
}

float* readFile(char* filename) {
	ifstream file;
	int i = 0;
	float coordinates;
	float* galaxies = new float[GALAXY_COUNT];
	file.open(filename);
	if (!file) {
		printf("Error opening the file");
		exit(1);
	}
	while (file >> coordinates)
	{
		// Converting from arch minutes to radians.
		galaxies[i] = coordinates * conversionFactor;
		++i;
	}
	file.close();
	return galaxies;
}

int main() {
	printf("%f", conversionFactor);
	int arraySize = (GALAXY_COUNT / 2) * sizeof(float);
	float* galaxyAnglesR = new float[arraySize * arraySize];
	float* galaxyAnglesS = new float[arraySize * arraySize];
	float* galaxyAnglesRS = new float[arraySize * arraySize];
	float* h_realGalaxies = readFile("data_100k_arcmin.txt");
	float* h_syntheticGalaxies = readFile("flat_100k_arcmin.txt");
	float* h_RdotProducts = new float[arraySize * arraySize];
	float* h_SdotProducts = new float[arraySize * arraySize];
	float* h_RSdotProducts = new float[arraySize * arraySize];
	float* d_realGalaxies; hipMalloc(&d_realGalaxies, arraySize);
	float* d_syntheticGalaxies; hipMalloc(&d_syntheticGalaxies, arraySize);
	float* d_RdotProducts; hipMalloc(&d_RdotProducts, arraySize);
	float* d_SdotProducts; hipMalloc(&d_SdotProducts, arraySize);
	float* d_RSdotProducts; hipMalloc(&d_RSdotProducts, arraySize);
	float* histogramBinsRR = new float[720];
	float* histogramBinsSS = new float[720];
	float* histogramBinsRS = new float[720];
	// Intializing the CUDA computation kernel
	int threadsInBlock = 256;
	int blocksInGrid = 100;
	hipMemcpy(d_realGalaxies, h_realGalaxies, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_syntheticGalaxies, h_syntheticGalaxies, arraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_RdotProducts, h_RdotProducts, arraySize, hipMemcpyHostToDevice);
	// Execute the function on GPU
	calculateAngles <<<blocksInGrid, threadsInBlock>>> (d_realGalaxies, d_syntheticGalaxies, d_RdotProducts, d_SdotProducts);
	hipMemcpy(h_RdotProducts, d_RdotProducts, arraySize, hipMemcpyDeviceToHost);
	hipMemcpy(h_SdotProducts, d_SdotProducts, arraySize, hipMemcpyDeviceToHost);
	hipMemcpy(h_RSdotProducts, d_RSdotProducts, arraySize, hipMemcpyDeviceToHost);
	for (int i = 0; i < 1000000; i++)
	{
		galaxyAnglesR[i] = acos(h_RdotProducts[i]) * (180 / pi);
	}
	float increment = 0.0;
	for (int i = 0; i < 720; i++)
	{
		for (int j = 0; j < 1000000; j++) 
		{
			if (galaxyAnglesR[j] <= increment && galaxyAnglesR[j] > increment - 0.25) {
				histogramBinsRR[i] += 1;
			}
			if (galaxyAnglesS[j] <= increment && galaxyAnglesS[j] > increment - 0.25) {
				histogramBinsSS[i] += 1;
			}
			if (galaxyAnglesRS[j] <= increment && galaxyAnglesRS[j] > increment - 0.25) {
				histogramBinsRS[i] += 1;
			}
		}
		increment += 0.25;
	}
	for (int i = 0; i < 720; i++) {
		printf("%f \n", histogramBinsRR[i]);
	}
}